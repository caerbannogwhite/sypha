
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>

#define CHECK(call) \
{ \
	const hipError_t error = call; \
	if (error != hipSuccess) \
	{ \
		printf("Error: %s:%d, ", __FILE__, __LINE__); \
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
		exit(1); \
	} \
}


/*
#define DEBUG_SYNC __syncthreads();

__device__ void warp_reduce_max(float smem[64])
{

	smem[threadIdx.x] = smem[threadIdx.x+32] > smem[threadIdx.x] ? 
						smem[threadIdx.x+32] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+16] > smem[threadIdx.x] ? 
						smem[threadIdx.x+16] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+8] > smem[threadIdx.x] ? 
						smem[threadIdx.x+8] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+4] > smem[threadIdx.x] ? 
						smem[threadIdx.x+4] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+2] > smem[threadIdx.x] ? 
						smem[threadIdx.x+2] : smem[threadIdx.x]; DEBUG_SYNC;

	smem[threadIdx.x] = smem[threadIdx.x+1] > smem[threadIdx.x] ? 
						smem[threadIdx.x+1] : smem[threadIdx.x]; DEBUG_SYNC;

}*/

double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp,NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

__global__ void array_mult_dev(double *A, double *B, double *C) {
	int i = threadIdx.x;
	C[i] = A[i] * B[i];
}

void array_mult_host(double *A, double *B, double *C, const int N) {
	for (int idx = 0; idx < N; idx++) {
		C[idx] = A[idx] * B[idx];
	}
}

void array_mult_host_test(double *d_A, double *d_B, double *d_C, const int N) {
	double alpha, beta;
	for (int j = 0; j < N; ++j)
    {
        hipMemcpy(&alpha, &d_A[j], sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&beta, &d_B[j], sizeof(double), hipMemcpyDeviceToHost);
        alpha = - (alpha * beta);
        hipMemcpy(&d_C[j], &alpha, sizeof(double), hipMemcpyHostToDevice);
    }
}

void initialData(double *ip,int size) {
	// generate different seed for random number
	time_t t;
	srand((unsigned int) time(&t));
	for (int i=0; i<size; i++) {
		ip[i] = (double)( rand() & 0xFF )/10.0;
	}
}

bool checkResult(double *h_res, double *d_res, int n)
{
	double buff;
	for (int i = 0; i < n; ++i)
	{
		hipMemcpy(&buff, &d_res[i], sizeof(double), hipMemcpyDeviceToHost);
		if (abs(buff - h_res[i]) >= 1.E-12)
		{
			printf("Test failed on i=%d, host: %lf, dev: %lf\n", i, h_res[i], buff);
			return false;
		}
		//printf("%lf, %lf\n", h_res[i], buff);
	}
	printf("Test pass\n");
	return true;
}

int main(int argc, char **argv) {
	int repeat = 20;
	int nElem = 32*32;
	size_t nBytes = nElem * sizeof(double);
	
	double tStart, tEnd, sum;

	double *h_A, *h_B, *h_C;
	double *d_A, *d_B, *d_C;

	h_A = (double *)malloc(nBytes);
	h_B = (double *)malloc(nBytes);
	h_C = (double *)malloc(nBytes);

	hipMalloc((void **)&d_A, nBytes);
	hipMalloc((void **)&d_B, nBytes);
	hipMalloc((void **)&d_C, nBytes);
	
	std::cout << "Initializing data" << std::endl;
	initialData(h_A, nElem);
	initialData(h_B, nElem);

	sum = 0;
	for (int i = 0; i < 5; ++i)
	{
		tStart = cpuSecond();
		array_mult_host(h_A, h_B, h_C, nElem);
		tEnd = cpuSecond();
		sum += (tEnd - tStart) * 1000;
	}
	std::cout << "array_mult_host time " << (sum / repeat) << " ms" << std::endl;

	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
	
	// sum = 0;
	// for (int i = 0; i < repeat; ++i)
	// {
	// 	tStart = cpuSecond();
	// 	array_mult_host_test(h_A, h_B, h_C, nElem);
	// 	tEnd = cpuSecond();
	// 	sum += (tEnd - tStart) * 1000;
	// }
	// std::cout << "array_mult_host_test time " << (sum / repeat) << " ms" << std::endl; 
	
	sum = 0;
	for (int i = 0; i < repeat; ++i)
	{
		tStart = cpuSecond();
		array_mult_dev<<<1, 32>>>(d_A, d_B, d_C);
		hipDeviceSynchronize();
		tEnd = cpuSecond();
		sum += (tEnd - tStart) * 1000;
	}
	std::cout << "array_mult_dev time " << (sum / repeat) << " ms" << std::endl; 
	

	checkResult(h_C, d_C, nElem);

	free(h_A);
	free(h_B);
	free(h_C);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	return(0);
}
