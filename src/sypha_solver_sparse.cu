#include "hip/hip_runtime.h"

#include "sypha_solver_sparse.h"

SyphaStatus solver_sparse_mehrotra(SyphaNodeSparse &node)
{
    const int reorder = 0;
    int singularity = 0;

    int i = 0, j = 0, k = 0, iterations = 0;
    size_t bufferSize = 0;
    size_t currBufferSize = 0;
    double alpha, beta, alphaPrim, alphaDual, sigma, mu, muAff;
    double alphaMaxPrim, alphaMaxDual;
    double *d_bufferX = NULL;
    double *d_bufferS = NULL;
    double *d_buffer = NULL;
    char message[1024];

    hipsparseMatDescr_t A_descr;

    ///////////////////             GET TRANSPOSED MATRIX
    
    // checkCudaErrors(hipMalloc((void **)&node.d_csrMatTransOffs, sizeof(int) * (node.ncols + 1)));
    // checkCudaErrors(hipMalloc((void **)&node.d_csrMatTransInds, sizeof(int) * node.nnz));
    // checkCudaErrors(hipMalloc((void **)&node.d_csrMatTransVals, sizeof(double) * node.nnz));

    // checkCudaErrors(hipDeviceSynchronize());

    // checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
    //                                               node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
    //                                               node.d_csrMatTransVals, node.d_csrMatTransOffs, node.d_csrMatTransInds,
    //                                               HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
    //                                               HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
    //                                               &bufferSize));

    // checkCudaErrors(hipMalloc((void **)&d_buffer, bufferSize));

    // checkCudaErrors(hipsparseCsr2cscEx2(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
    //                                    node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
    //                                    node.d_csrMatTransVals, node.d_csrMatTransOffs, node.d_csrMatTransInds,
    //                                    HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
    //                                    HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
    //                                    d_buffer));

    // checkCudaErrors(hipsparseCreateCsr(&node.matTransDescr, node.ncols, node.nrows, node.nnz,
    //                                   //node.d_csrMatTransVals, node.d_csrMatTransOffs, node.d_csrMatTransInds,
    //                                   node.d_csrMatTransOffs, node.d_csrMatTransInds, node.d_csrMatTransVals,
    //                                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    //                                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    ///////////////////             GET STARTING POINT
    // initialise x, y, s
    node.h_x = (double *)malloc(sizeof(double) * node.ncols);
    node.h_y = (double *)malloc(sizeof(double) * node.nrows);
    node.h_s = (double *)malloc(sizeof(double) * node.ncols);

    node.timeStartSolStart = node.env->timer();
    solver_sparse_mehrotra_init_gsl(node);
    node.timeStartSolEnd = node.env->timer();

    ///////////////////             SET BIG MATRIX ON HOST
    //
    // On each step we solve this linear system twice:
    //
    //      O | A' | I    x    -rc
    //      --|----|---   -    ---
    //      A | O  | O  * y  = -rb
    //      --|----|---   -    ---
    //      S | O  | X    s    -rxs
    //
    // Where A is the model matrix (standard form), I is the n*n identity
    // matrix, S is the n*n s diagonal matrix, X is the n*n diagonal matrix.
    // Total number of non-zero elements is A.nnz * 2 + n * 3

    node.timePreSolStart = node.env->timer();

    int A_nrows = node.ncols * 2 + node.nrows;
    int A_ncols = A_nrows;
    int A_nnz = node.nnz * 2 + node.ncols * 3;
    
    int *h_csrAInds = NULL;
    int *h_csrAOffs = NULL;
    double *h_csrAVals = NULL;

    int *d_csrAInds = NULL;
    int *d_csrAOffs = NULL;
    double *d_csrAVals = NULL;

    double *d_rhs = NULL;
    double *d_sol = NULL;
    double *d_prevSol = NULL;

    h_csrAInds = (int *)calloc(sizeof(int), A_nnz);
    h_csrAOffs = (int *)calloc(sizeof(int), (A_nrows + 1));
    h_csrAVals = (double *)calloc(sizeof(double), A_nnz);

    sprintf(message, "Initialising matrix: %d rows, %d columns, %d non zeros", A_nrows, A_ncols, A_nnz);
    node.env->logger(message, "INFO", 17);

    // Instantiate the first group of n rows: O | A' | I
    bool found = false;
    int off = 0, rowCnt = 0;

    h_csrAOffs[0] = 0;
    for (j = 0; j < node.ncols; ++j)
    {
        rowCnt = 0;
        for (i = 0; i < node.nrows; ++i)
        {
            found = false;
            for (k = node.h_csrMatOffs->data()[i]; k < node.h_csrMatOffs->data()[i+1]; ++k)
            {
                if (node.h_csrMatInds->data()[k] == j)
                {
                    found = true;
                    break;
                }
            }

            if (found)
            {
                h_csrAInds[off] = node.ncols + i;
                h_csrAVals[off] = node.h_csrMatVals->data()[k];
                ++rowCnt;
                ++off;
            }
        }

        // append the I matrix element for the current row
        h_csrAInds[off] = node.ncols + node.nrows + j;
        h_csrAVals[off] = 1.0;
        ++rowCnt;
        ++off;

        h_csrAOffs[j + 1] = h_csrAOffs[j] + rowCnt;
    }

    // Instantiate the second group of m rows: A | O | O
    for (i = 0; i < node.nrows; ++i)
    {
        h_csrAOffs[node.ncols + i + 1] = h_csrAOffs[node.ncols + i] + (node.h_csrMatOffs->data()[i + 1] - node.h_csrMatOffs->data()[i]);
    }
    memcpy(&h_csrAInds[off], node.h_csrMatInds->data(), sizeof(int) * node.nnz);
    memcpy(&h_csrAVals[off], node.h_csrMatVals->data(), sizeof(double) * node.nnz);
    off += node.nnz;

    // Instantiate the third group of n rows: S | O | X
    for (j = 0; j < node.ncols; ++j)
    {
        // s
        h_csrAInds[off] = j;
        h_csrAVals[off] = node.h_s[j];
        ++off;

        // x
        h_csrAInds[off] = node.ncols + node.nrows + j;
        h_csrAVals[off] = node.h_x[j];
        ++off;

        h_csrAOffs[node.ncols + node.nrows + j + 1] = h_csrAOffs[node.ncols + node.nrows + j] + 2;
    }

    checkCudaErrors(hipMalloc((void **)&d_csrAInds, sizeof(int) * A_nnz));
    checkCudaErrors(hipMalloc((void **)&d_csrAOffs, sizeof(int) * (A_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&d_csrAVals, sizeof(double) * A_nnz));

    checkCudaErrors(hipMemcpy(d_csrAInds, h_csrAInds, sizeof(int) * A_nnz, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrAOffs, h_csrAOffs, sizeof(int) * (A_nrows + 1), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_csrAVals, h_csrAVals, sizeof(double) * A_nnz, hipMemcpyHostToDevice));
    
    checkCudaErrors(hipsparseCreateMatDescr(&A_descr));
    checkCudaErrors(hipsparseSetMatType(A_descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(A_descr, HIPSPARSE_INDEX_BASE_ZERO));
    
    ///////////////////             TEST
    // double *d_ADn = NULL;
    // checkCudaErrors(hipMalloc((void **)&d_ADn, sizeof(double) * A_nrows * A_ncols));

    // checkCudaErrors(hipsparseDcsr2dense(node.cusparseHandle, A_nrows, A_ncols,
    //                                    A_descr, // HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO
    //                                    d_csrAVals, d_csrAOffs, d_csrAInds,
    //                                    d_ADn, A_nrows));

    // utils_printDmat(A_nrows, A_ncols, A_nrows, d_ADn, true);
    // checkCudaErrors(hipFree(d_ADn));

    // printf("OFFS:\n");
    // utils_printIvec(A_nrows+1, d_csrAOffs, true);
    // printf("INDS:\n");
    // utils_printIvec(A_nnz, d_csrAInds, true);
    // printf("VALS:\n");
    // utils_printDvec(A_nnz, d_csrAVals, true);
    ///////////////////             END TEST

    free(h_csrAInds);
    free(h_csrAOffs);
    free(h_csrAVals);

    ///////////////////             INITIALISE RHS
    
    node.env->logger("Initialise right-hand-side", "INFO", 17);
    checkCudaErrors(hipMalloc((void **)&d_rhs, sizeof(double) * A_nrows));
    checkCudaErrors(hipMalloc((void **)&d_sol, sizeof(double) * A_nrows));
    checkCudaErrors(hipMalloc((void **)&d_prevSol, sizeof(double) * A_nrows));

    // put x, y, s on device sol as [x, y, s]
    double *d_x = d_prevSol;
    double *d_y = &d_prevSol[node.ncols];
    double *d_s = &d_prevSol[node.ncols + node.nrows];

    double *d_deltaX = d_sol;
    double *d_deltaY = &d_sol[node.ncols];
    double *d_deltaS = &d_sol[node.ncols + node.nrows];

    checkCudaErrors(hipMemcpy(d_x, node.h_x, sizeof(double) * node.ncols, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, node.h_y, sizeof(double) * node.nrows, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_s, node.h_s, sizeof(double) * node.ncols, hipMemcpyHostToDevice));

    // put OBJ and S on device rhs
    double *d_resC = d_rhs;
    double *d_resB = &d_rhs[node.ncols];
    double *d_resXS = &d_rhs[node.ncols + node.nrows];

    checkCudaErrors(hipMemcpy(d_resC, node.d_ObjDns, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemcpy(d_resB, node.d_RhsDns, sizeof(double) * node.nrows, hipMemcpyDeviceToDevice));

    // Residuals
    // resB, resC equation 14.7, page 395(414)Numerical Optimization
    // resC = -mat' * y + (obj - s)
    // resB = -mat  * x + rhs

    hipsparseDnVecDescr_t vecX, vecY, vecResC, vecResB;

    checkCudaErrors(hipsparseCreateDnVec(&vecX, (int64_t)node.ncols, d_x, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecY, (int64_t)node.nrows, d_y, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecResC, (int64_t)node.ncols, d_resC, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecResB, (int64_t)node.nrows, d_resB, HIP_R_64F));

    alpha = -1.0;
    checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                &alpha, d_s, 1, d_resC, 1));

    alpha = -1.0;
    beta = 1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                                            &alpha, node.matDescr, vecY,
                                            &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize));

    // buffer size for other needs
    currBufferSize = (size_t)(sizeof(double) * node.ncols * 2);
    currBufferSize = currBufferSize > bufferSize ? currBufferSize : bufferSize;
    checkCudaErrors(hipMalloc((void **)&d_buffer, currBufferSize));

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle, HIPSPARSE_OPERATION_TRANSPOSE,
                                 &alpha, node.matDescr, vecY,
                                 &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 d_buffer));

    alpha = -1.0;
    beta = 1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, node.matDescr, vecX,
                                            &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize));

    if (bufferSize > currBufferSize)
    {
        currBufferSize = bufferSize;
        checkCudaErrors(hipMalloc((void **)&d_buffer, currBufferSize));
    }

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, node.matDescr, vecX,
                                 &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 (size_t *)d_buffer));

    ///////////////////             CALCULATE MU
    // duality measure, defined at page 395(414) Numerical Optimization
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
    mu /= node.ncols;

    node.timePreSolEnd = node.env->timer();

    ///////////////////             MAIN LOOP

    node.env->logger("Starting Mehrotra proceduce", "INFO", 17);
    node.timeSolverStart = node.env->timer();
    while ((iterations < node.env->MEHROTRA_MAX_ITER) && (mu > node.env->MEHROTRA_MU_TOL))
    {


        // x, s multiplication and res XS update: to improve
        //elem_min_mult_hybr(d_x, d_s, d_resXS, node.ncols);
        elem_min_mult_dev(d_x, d_s, d_resXS, node.ncols);
        
        checkCudaErrors(hipsolverSpDcsrlsvqr(node.cusolverSpHandle,
                                            A_nrows, A_nnz, A_descr,
                                            d_csrAVals, d_csrAOffs, d_csrAInds,
                                            d_rhs,
                                            node.env->MEHROTRA_CHOL_TOL, reorder,
                                            d_sol, &singularity));

        ///////////////             TEST
        /*printf("\n%4d) AFTER AFFINE SYSTEM\n", iterations);
        double *d_ADn = NULL;
        checkCudaErrors(hipMalloc((void **)&d_ADn, sizeof(double) * A_nrows * A_ncols));

        checkCudaErrors(hipsparseDcsr2dense(node.cusparseHandle, A_nrows, A_ncols,
                                           A_descr, // HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO
                                           d_csrAVals, d_csrAOffs, d_csrAInds,
                                           d_ADn, A_nrows));

        utils_printDmat(A_nrows, A_ncols, A_nrows, d_ADn, true, true);
        checkCudaErrors(hipFree(d_ADn));

        printf("sol:\n");
        utils_printDvec(node.ncols * 2 + node.nrows, d_sol, true);
        printf("rhs:\n");
        utils_printDvec(node.ncols * 2 + node.nrows, d_rhs, true);*/
        ///////////////             END TEST

        // affine step length, definition 14.32 at page 408(427)
        // alpha_max_p = min([-xi / delta_xi for xi, delta_xi in zip(x, delta_x_aff) if delta_xi < 0.0])
        // alpha_max_d = min([-si / delta_si for si, delta_si in zip(s, delta_s_aff) if delta_si < 0.0])

        // finding alphaMaxPrim and alphaMaxDual: to improve
        find_alpha_max(&alphaMaxPrim, &alphaMaxDual,
                       d_x, d_deltaX, d_s, d_deltaS, node.ncols);

        alphaPrim = gsl_min(1.0, alphaMaxPrim);
        alphaDual = gsl_min(1.0, alphaMaxDual);

        // mu_aff = (x + alpha_aff_p * delta_x_aff).dot(s + alpha_aff_d * delta_s_aff) / float(n)
        // d_deltaX, d_deltaY, d_deltaS are pointees to d_sol
        // the solution of the previous system
        // the dimension of the buffer is guaranteed to be >= 2 * ncols
        d_bufferX = d_buffer;
        d_bufferS = &d_buffer[node.ncols];
        checkCudaErrors(hipMemcpyAsync(d_bufferX, d_x, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));
        checkCudaErrors(hipMemcpyAsync(d_bufferS, d_s, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaPrim, d_deltaX, 1, d_bufferX, 1));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaDual, d_deltaS, 1, d_bufferS, 1));

        checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_bufferX, 1, d_bufferS, 1, &muAff));
        muAff /= node.ncols;

        // corrector step or centering parameter
        sigma = gsl_pow_3(muAff / mu);

        ///////////////             TEST
        // printf("\n\n%4d) PRE CORRECTION SYSTEM\n", iterations);
        // printf("sigma: %lf, muAff: %lf\n", sigma, muAff);
        // printf("d buff X:\n");
        // utils_printDvec(node.ncols, d_bufferX, true);
        // printf("d buff S:\n");
        // utils_printDvec(node.ncols, d_bufferS, true);
        ///////////////             END TEST

        // x, s multiplication and res XS update: to improve
        for (j = 0; j < node.ncols; ++j)
        {
            checkCudaErrors(hipMemcpyAsync(&alpha, &d_deltaX[j], sizeof(double), hipMemcpyDeviceToHost, node.cudaStream));
            checkCudaErrors(hipMemcpyAsync(&beta, &d_deltaS[j], sizeof(double), hipMemcpyDeviceToHost, node.cudaStream));
            alpha = -(alpha * beta) + sigma * mu;
            checkCudaErrors(hipMemcpyAsync(&d_bufferX[j], &alpha, sizeof(double), hipMemcpyHostToDevice, node.cudaStream));
        }

        alpha = 1.0;
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alpha, d_bufferX, 1, d_resXS, 1));


        checkCudaErrors(hipsolverSpDcsrlsvqr(node.cusolverSpHandle,
                                            A_nrows, A_nnz, A_descr,
                                            d_csrAVals, d_csrAOffs, d_csrAInds,
                                            d_rhs,
                                            node.env->MEHROTRA_CHOL_TOL, reorder,
                                            d_sol, &singularity));
                                            
        ///////////////             TEST
        // printf("\n%4d) AFTER CORRECTION SYSTEM\n", iterations);
        // printf("sol:\n");
        // utils_printDvec(node.ncols * 2 + node.nrows, d_sol, true);
        // printf("rhs:\n");
        // utils_printDvec(node.ncols * 2 + node.nrows, d_rhs, true);
        ///////////////             END TEST

        // finding alphaMaxPrim and alphaMaxDual: to improve
        // finding alphaMaxPrim and alphaMaxDual: to improve
        find_alpha_max(&alphaMaxPrim, &alphaMaxDual,
                       d_x, d_deltaX, d_s, d_deltaS, node.ncols);

        alphaPrim = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxPrim);
        alphaDual = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxDual);

        // d_deltaX, d_deltaY, d_deltaS are pointees to d_sol
        // the solution of the previous system 

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaPrim, d_deltaX, 1, d_x, 1));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.nrows,
                                    &alphaDual, d_deltaY, 1, d_y, 1));
        
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaDual, d_deltaS, 1, d_s, 1));

        ///////////////             UPDATE

        alpha = -(alphaDual - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.ncols,
                                    &alpha, d_resC, 1));

        alpha = -(alphaPrim - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.nrows,
                                    &alpha, d_resB, 1));


        checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
        mu /= node.ncols;
        
        ///////////////             TEST
        // printf("\n%4d) UPDATE STEP\n", iterations);
        // printf("mu: %8.6lf, al prim: %8.6lf, al max prim: %8.6lf, al dual: %8.6lf, al max dual: %8.6lf\n", mu, alphaPrim, alphaMaxPrim, alphaDual, alphaMaxDual);
        ///////////////             END TEST

        // update x and s on matrix
        off = node.nnz * 2 + node.ncols;
        checkCudaErrors(hipblasDcopy(node.cublasHandle, node.ncols, d_s, 1, &d_csrAVals[off], 2));
        checkCudaErrors(hipblasDcopy(node.cublasHandle, node.ncols, d_x, 1, &d_csrAVals[off + 1], 2));

        ++iterations;

        ///////////////             TEST
        //printf("\n\nLOOP END\n");
        //printf("al prim: %lf, al dual: %lf, mu: %lf\n", alphaPrim, alphaDual, mu);
        //printf("X:\n");
        //utils_printDvec(node.ncols, d_x, true);
        //printf("Y:\n");
        //utils_printDvec(node.nrows, d_y, true);
        //printf("S:\n");
        //utils_printDvec(node.ncols, d_s, true);
        //printf("delta X:\n");
        //utils_printDvec(node.ncols, d_deltaX, true);
        //printf("delta S:\n");
        //utils_printDvec(node.ncols, d_deltaS, true);
        ///////////////             END TEST
    }

    node.iterations = iterations;
    
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols,
                               d_x, 1, node.d_ObjDns, 1, &node.objvalPrim));

    checkCudaErrors(hipblasDdot(node.cublasHandle, node.nrows,
                               d_y, 1, node.d_RhsDns, 1, &node.objvalDual));

    node.env->logger("Mehrotra procedure complete", "INFO", 10);
    node.timeSolverEnd = node.env->timer();

    ///////////////////             RELEASE RESOURCES

    checkCudaErrors(hipsparseDestroyMatDescr(A_descr));

    checkCudaErrors(hipFree(d_csrAInds));
    checkCudaErrors(hipFree(d_csrAOffs));
    checkCudaErrors(hipFree(d_csrAVals));

    checkCudaErrors(hipFree(d_rhs));
    checkCudaErrors(hipFree(d_sol));
    checkCudaErrors(hipFree(d_prevSol));

    checkCudaErrors(hipsparseDestroyDnVec(vecX));
    checkCudaErrors(hipsparseDestroyDnVec(vecY));
    checkCudaErrors(hipsparseDestroyDnVec(vecResC));
    checkCudaErrors(hipsparseDestroyDnVec(vecResB));

    checkCudaErrors(hipsparseDestroySpMat(node.matTransDescr));
    node.matTransDescr = NULL;

    // checkCudaErrors(hipFree(node.d_csrMatTransInds));
    // checkCudaErrors(hipFree(node.d_csrMatTransOffs));
    // checkCudaErrors(hipFree(node.d_csrMatTransVals));

    // node.d_csrMatTransInds = NULL;
    // node.d_csrMatTransOffs = NULL;
    // node.d_csrMatTransVals = NULL;

    if (d_buffer) checkCudaErrors(hipFree(d_buffer));

    return CODE_SUCCESFULL;
}

SyphaStatus solver_sparse_mehrotra_2(SyphaNodeSparse &node)
{
    const int reorder = 0;
    int singularity = 0;

    int i = 0, j = 0, k = 0, iterations = 0;
    double alpha, beta, alphaPrim, alphaDual, sigma, mu, muAff;
    double alphaMaxPrim, alphaMaxDual;
    double *d_bufferX = NULL;
    double *d_bufferS = NULL;

    double *d_resC = NULL, *d_resB = NULL, *d_resXS = NULL;
    double *d_tmpA = NULL, *d_tmpB = NULL;
    double *d_x = NULL, *d_y = NULL, *d_s = NULL, *d_invS = NULL;
    double *d_delX = NULL, *d_delY = NULL, *d_delS = NULL;

    char message[1024];

    hipsparseDnVecDescr_t vecX, vecY, vecS, vecResC, vecResB;
    hipsparseDnVecDescr_t vecDelX, vecDelY, vecDelS, vecTmpA, vecTmpB;
    hipsparseSpGEMMDescr_t spgemmDescr;

    hipsparseMatDescr_t matDescrGen;
    checkCudaErrors(hipsparseCreateMatDescr(&matDescrGen));
    checkCudaErrors(hipsparseSetMatType(matDescrGen, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(matDescrGen, HIPSPARSE_INDEX_BASE_ZERO));

    // AT
    int64_t AT_nrows = node.ncols, AT_ncols = node.nrows, AT_nnz = node.nnz;
    int *d_AToffs = NULL, *d_ATinds = NULL;
    double *d_ATvals = NULL;
    hipsparseSpMatDescr_t AT_descr;

    checkCudaErrors(hipMalloc((void **)&d_AToffs, sizeof(int) * (AT_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&d_ATinds, sizeof(int) * AT_nnz));
    checkCudaErrors(hipMalloc((void **)&d_ATvals, sizeof(double) * AT_nnz));

    checkCudaErrors(hipsparseCreateCsr(&AT_descr, AT_nrows, AT_ncols, AT_nnz,
                                      d_AToffs, d_ATinds, d_ATvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // D
    int64_t D_nrows = node.ncols, D_ncols = node.ncols, D_nnz = node.ncols;
    int *d_Doffs = NULL, *d_Dinds = NULL;
    double *d_Dvals = NULL;
    hipsparseSpMatDescr_t D_descr;

    checkCudaErrors(hipMalloc((void **)&d_Doffs, sizeof(int) * (D_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&d_Dinds, sizeof(int) * D_nrows));
    checkCudaErrors(hipMalloc((void **)&d_Dvals, sizeof(double) * D_nrows));

    checkCudaErrors(hipsparseCreateCsr(&D_descr, D_nrows, D_ncols, D_nnz,
                                      d_Doffs, d_Dinds, d_Dvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // AD
    int64_t AD_nrows = node.nrows, AD_ncols = node.ncols, AD_nnz = 0, AD_currNnz = 0;
    int *d_ADoffs = NULL, *d_ADinds = NULL;
    double *d_ADvals = NULL;
    hipsparseSpMatDescr_t AD_descr;

    checkCudaErrors(hipMalloc((void **)&d_ADoffs, sizeof(int) * (AD_nrows + 1)));

    checkCudaErrors(hipsparseCreateCsr(&AD_descr, AD_nrows, AD_ncols, AD_nnz,
                                      d_ADoffs, d_ADinds, d_ADvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // ADA
    int64_t ADA_nrows = node.nrows, ADA_ncols = node.nrows, ADA_nnz = 0, ADA_currNnz = 0;
    int *d_ADAoffs = NULL, *d_ADAinds = NULL;
    double *d_ADAvals = NULL;
    hipsparseSpMatDescr_t ADA_descr;

    checkCudaErrors(hipMalloc((void **)&d_ADAoffs, sizeof(int) * (ADA_nrows + 1)));
    
    checkCudaErrors(hipsparseCreateCsr(&ADA_descr, ADA_nrows, ADA_ncols, ADA_nnz,
                                      d_ADAoffs, d_ADAinds, d_ADAvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // BUFFER
    size_t bufferSize1 = 0;
    size_t currBufferSize1 = (size_t)(sizeof(double) * node.ncols * 2);
    double *d_buffer1 = NULL;

    size_t bufferSize2 = 0;
    size_t currBufferSize2 = 0;
    double *d_buffer2 = NULL;

    checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));

    ///////////////////             GET TRANSPOSED MATRIX

    checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
                                                  node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
                                                  d_ATvals, d_AToffs, d_ATinds,
                                                  HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
                                                  HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
                                                  &bufferSize1));
    
    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseCsr2cscEx2(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
                                       node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
                                       d_ATvals, d_AToffs, d_ATinds,
                                       HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
                                       HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
                                       d_buffer1));

    ///////////////////             GET STARTING POINT
    // initialise x, y, s
    node.h_x = (double *)malloc(sizeof(double) * node.ncols);
    node.h_y = (double *)malloc(sizeof(double) * node.nrows);
    node.h_s = (double *)malloc(sizeof(double) * node.ncols);

    node.timeStartSolStart = node.env->timer();
    solver_sparse_mehrotra_init_gsl(node);
    node.timeStartSolEnd = node.env->timer();

    ///////////////////             INITIALISE RHS

    node.env->logger("Initialise right-hand-side", "INFO", 17);
    node.timePreSolStart = node.env->timer();

    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_y, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_s, sizeof(double) * node.ncols));

    checkCudaErrors(hipMalloc((void **)&d_delX, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_delY, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_delS, sizeof(double) * node.ncols));

    checkCudaErrors(hipMemcpyAsync(d_x, node.h_x, sizeof(double) * node.ncols, hipMemcpyHostToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_y, node.h_y, sizeof(double) * node.nrows, hipMemcpyHostToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_s, node.h_s, sizeof(double) * node.ncols, hipMemcpyHostToDevice, node.cudaStream));

    // put OBJ and S on device rhs
    checkCudaErrors(hipMalloc((void **)&d_resC, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_resB, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_resXS, sizeof(double) * node.ncols));

    checkCudaErrors(hipMemcpyAsync(d_resC, node.d_ObjDns, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_resB, node.d_RhsDns, sizeof(double) * node.nrows, hipMemcpyDeviceToDevice, node.cudaStream));

    // Residuals
    // resB, resC equation 14.7, page 395(414)Numerical Optimization
    // resC = AT * Y - (obj - s)
    // resB = A  * X - rhs

    checkCudaErrors(hipsparseCreateDnVec(&vecX, (int64_t)node.ncols, d_x, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecY, (int64_t)node.nrows, d_y, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecS, (int64_t)node.ncols, d_s, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecDelX, (int64_t)node.ncols, d_delX, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecDelY, (int64_t)node.nrows, d_delY, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecDelS, (int64_t)node.ncols, d_delS, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecResC, (int64_t)node.ncols, d_resC, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecResB, (int64_t)node.nrows, d_resB, HIP_R_64F));

    alpha = -1.0;
    checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                &alpha, d_s, 1, d_resC, 1));

    alpha = 1.0;
    beta = -1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, AT_descr, vecY,
                                            &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize1));

    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, AT_descr, vecY,
                                 &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 d_buffer1));

    alpha = 1.0;
    beta = -1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, node.matDescr, vecX,
                                            &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize1));

    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, node.matDescr, vecX,
                                 &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 (size_t *)d_buffer1));

    ///////////////////             CALCULATE MU
    // duality measure, defined at page 395(414) Numerical Optimization
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
    mu /= node.ncols;

    node.timePreSolEnd = node.env->timer();

    ///////////////////             SET UP D AND INV(S)
    
    int bSize = (node.ncols >> 5) + 1;

    checkCudaErrors(hipMalloc((void **)&d_invS, sizeof(double) * node.ncols));
    
    hipDeviceSynchronize();
    range_kernel<<<bSize, 32>>>(d_Doffs, node.ncols + 1);
    range_kernel<<<bSize, 32>>>(d_Dinds, node.ncols);
    
    ///////////////////             MAIN LOOP
    
    node.env->logger("Starting Mehrotra proceduce", "INFO", 17);
    node.timeSolverStart = node.env->timer();
    
    iterations = 0;
    while ((iterations < node.env->MEHROTRA_MAX_ITER) && (mu > node.env->MEHROTRA_MU_TOL))
    {
        // x, s multiplication and res XS update: to improve
        hipDeviceSynchronize();
        elem_mult_kernel<<<bSize, 32>>>(d_x, d_s, d_resXS, node.ncols);

        elem_inv_kernel<<<bSize, 32>>>(d_s, d_invS, node.ncols);

        hipDeviceSynchronize();
        elem_mult_kernel<<<bSize, 32>>>(d_x, d_invS, d_Dvals, node.ncols);

        // hipDeviceSynchronize();
        // printf("%d) START\n", iterations);
        // printf("X\n");
        // utils_printDvec(node.ncols, d_x, true);
        // printf("INV(S)\n");
        // utils_printDvec(node.ncols, d_invS, true);
        // printf("D\n");
        // utils_printDvec(node.ncols, d_Dvals, true);
        // printf("RES B\n");
        // utils_printDvec(node.nrows, d_resB, true);
        // printf("RES C\n");
        // utils_printDvec(node.ncols, d_resC, true);

        ///////////////             COMPUTE AD
        {
            alpha = 1.0;
            beta = 0.0;

            // SpGEMM Computation
            checkCudaErrors(hipsparseSpGEMM_createDescr(&spgemmDescr));

            // ask bufferSize1 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, D_descr,
                                                  &beta, AD_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, NULL));

            if (bufferSize1 > currBufferSize1)
            {
                currBufferSize1 = bufferSize1;
                if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
                checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
            }

            // inspect the matrices D and AT to understand the memory requiremnent for
            // the next step
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, D_descr,
                                                  &beta, AD_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, d_buffer1));

            // ask bufferSize2 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, D_descr,
                                           &beta, AD_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, NULL));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            // compute the intermediate product of A * B
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, D_descr,
                                           &beta, AD_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, d_buffer2));

            // get matrix DA non-zero entries
            hipsparseSpMatGetSize(AD_descr, &AD_nrows, &AD_ncols, &AD_nnz);

            // allocate matrix DA
            if (AD_nnz > AD_currNnz)
            {
                AD_currNnz = AD_nnz;
                if (d_ADinds) checkCudaErrors(hipFree(d_ADinds));
                if (d_ADvals) checkCudaErrors(hipFree(d_ADvals));
                checkCudaErrors(hipMalloc((void **)&d_ADinds, sizeof(int) * AD_currNnz));
                checkCudaErrors(hipMalloc((void **)&d_ADvals, sizeof(double) * AD_currNnz));
            }

            // update DA with the new pointers
            checkCudaErrors(hipsparseCsrSetPointers(AD_descr, d_ADoffs, d_ADinds, d_ADvals));

            // copy the final products to the matrix AAT
            checkCudaErrors(hipsparseSpGEMM_copy(node.cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, node.matDescr, D_descr,
                                        &beta, AD_descr,
                                        HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDescr));

            checkCudaErrors(hipsparseSpGEMM_destroyDescr(spgemmDescr));
        }

        ///////////////             COMPUTE ADA
        {
            alpha = 1.0;
            beta = 0.0;

            // SpGEMM Computation
            checkCudaErrors(hipsparseSpGEMM_createDescr(&spgemmDescr));

            // ask bufferSize1 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, AD_descr, AT_descr,
                                                  &beta, ADA_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, NULL));

            if (bufferSize1 > currBufferSize1)
            {
                currBufferSize1 = bufferSize1;
                if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
                checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
            }

            // inspect the matrices A and DA to understand the memory requiremnent for
            // the next step
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, AD_descr, AT_descr,
                                                  &beta, ADA_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, d_buffer1));

            // ask bufferSize2 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, AD_descr, AT_descr,
                                           &beta, ADA_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, NULL));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            // compute the intermediate product of A * DA
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, AD_descr, AT_descr,
                                           &beta, ADA_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, d_buffer2));

            // get matrix ADA non-zero entries
            hipsparseSpMatGetSize(ADA_descr, &ADA_nrows, &ADA_ncols, &ADA_nnz);

            // allocate matrix ADA
            if (ADA_nnz > ADA_currNnz)
            {
                ADA_currNnz = ADA_nnz;
                if (d_ADAinds) checkCudaErrors(hipFree(d_ADAinds));
                if (d_ADAvals) checkCudaErrors(hipFree(d_ADAvals));
                checkCudaErrors(hipMalloc((void **)&d_ADAinds, sizeof(int) * ADA_currNnz));
                checkCudaErrors(hipMalloc((void **)&d_ADAvals, sizeof(double) * ADA_currNnz));
            }

            // update ADA with the new pointers
            checkCudaErrors(hipsparseCsrSetPointers(ADA_descr, d_ADAoffs, d_ADAinds, d_ADAvals));

            // copy the final products to the matrix ADA
            checkCudaErrors(hipsparseSpGEMM_copy(node.cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, AD_descr, AT_descr,
                                        &beta, ADA_descr,
                                        HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDescr));

            checkCudaErrors(hipsparseSpGEMM_destroyDescr(spgemmDescr));
        }

        ///////////////////             TEST
        // double *d_ADn = NULL;
        // checkCudaErrors(hipMalloc((void **)&d_ADn, sizeof(double) * ADA_nrows * ADA_ncols));

        // hipsparseMatDescr_t matGenDescr;
        // checkCudaErrors(hipsparseCreateMatDescr(&matGenDescr));
        // checkCudaErrors(hipsparseSetMatType(matGenDescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        // checkCudaErrors(hipsparseSetMatIndexBase(matGenDescr, HIPSPARSE_INDEX_BASE_ZERO));

        // checkCudaErrors(hipsparseDcsr2dense(node.cusparseHandle, ADA_nrows, ADA_ncols,
        //                                    matGenDescr, // HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO
        //                                    d_ADAvals, d_ADAoffs, d_ADAinds,
        //                                    d_ADn, ADA_nrows));

        // printf("%d) MAT\n", iterations);
        // utils_printDmat(ADA_nrows, ADA_ncols, ADA_nrows, d_ADn, true, false);
        // checkCudaErrors(hipFree(d_ADn));

        // printf("OFFS:\n");
        // utils_printIvec(A_nrows+1, d_csrAOffs, true);
        // printf("INDS:\n");
        // utils_printIvec(A_nnz, d_csrAInds, true);
        // printf("VALS:\n");
        // utils_printDvec(A_nnz, d_csrAVals, true);
        ///////////////////             END TEST

        ///////////////             COMPUTE TMPA = - AD * resC - resB

        // store TMPA vector on buffer 1, size of buffer 1 is guaranteed to 
        // be >= 2*ncols, copy resB on TMPA
        d_tmpA = d_buffer1;
        d_tmpB = &d_buffer1[node.ncols];
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpA, AD_nrows, d_tmpA, HIP_R_64F));
        checkCudaErrors(hipMemcpyAsync(d_tmpA, d_resB, sizeof(double) * AD_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        // compute TMPB
        elem_mult_kernel<<<bSize, 32>>>(d_resXS, d_invS, d_tmpB, node.ncols);
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpB, AD_ncols, d_tmpB, HIP_R_64F));

        {    
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE TMPA = TMPA + A * TMPB
        {    
            alpha = 1.0;
            beta = 1.0;

            hipDeviceSynchronize();

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE DEL_Y = SOLVE(ADA, TMPA)
        {
            checkCudaErrors(hipsolverSpDcsrlsvchol(node.cusolverSpHandle,
                                            ADA_nrows, ADA_nnz, matDescrGen,
                                            d_ADAvals, d_ADAoffs, d_ADAinds,
                                            d_tmpA,
                                            node.env->MEHROTRA_CHOL_TOL, reorder,
                                            d_delY, &singularity));
        }

        ///////////////             COMPUTE DEL_S = - AT * DEL_Y - resC
        checkCudaErrors(hipMemcpyAsync(d_delS, d_resC, sizeof(double) * AT_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        ///////////////             COMPUTE DEL_X = -TMP_B - D * DEL_S
        checkCudaErrors(hipMemcpyAsync(d_delX, d_tmpB, sizeof(double) * D_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        // printf("%d) AFFINE SYSTEM\n", iterations);
        // printf("delta X\n");
        // utils_printDvec(node.ncols, d_delX, true);
        // printf("delta Y\n");
        // utils_printDvec(node.nrows, d_delY, true);
        // printf("delta S\n");
        // utils_printDvec(node.ncols, d_delS, true);

        // affine step length, definition 14.32 at page 408(427)
        // alpha_max_p = min([-xi / delta_xi for xi, delta_xi in zip(x, delta_x_aff) if delta_xi < 0.0])
        // alpha_max_d = min([-si / delta_si for si, delta_si in zip(s, delta_s_aff) if delta_si < 0.0])

        // finding alphaMaxPrim and alphaMaxDual: to improve
        find_alpha_max(&alphaMaxPrim, &alphaMaxDual,
                       d_x, d_delX, d_s, d_delS, node.ncols);

        alphaPrim = gsl_min(1.0, alphaMaxPrim);
        alphaDual = gsl_min(1.0, alphaMaxDual);

        // mu_aff = (x + alpha_aff_p * delta_x_aff).dot(s + alpha_aff_d * delta_s_aff) / float(n)
        // d_deltaX, d_deltaY, d_deltaS are pointees to d_sol
        // the solution of the previous system
        // the dimension of the buffer is guaranteed to be >= 2 * ncols
        //d_tmpA = d_buffer1;
        //d_tmpB = &d_buffer1[node.ncols];
        checkCudaErrors(hipMemcpyAsync(d_tmpA, d_x, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));
        checkCudaErrors(hipMemcpyAsync(d_tmpB, d_s, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols, &alphaPrim, d_delX, 1, d_tmpA, 1));
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols, &alphaDual, d_delS, 1, d_tmpB, 1));

        checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_tmpA, 1, d_tmpB, 1, &muAff));
        muAff /= node.ncols;

        // corrector step or centering parameter
        sigma = gsl_pow_3(muAff / mu);

        elem_mult_kernel<<<bSize, 32>>>(d_delX, d_delS, d_tmpA, node.ncols);
        hipDeviceSynchronize();

        alpha = 1.0;
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols, &alpha, d_tmpA, 1, d_resXS, 1));

        alpha = - sigma * mu;
        scal_sum_kernel<<<bSize, 32>>>(alpha, d_resXS, node.ncols);
        hipDeviceSynchronize();

        ///////////////             COMPUTE TMPA = - AD * resC - resB

        // store TMPA vector on buffer 1, size of buffer 1 is guaranteed to 
        // be >= 2*ncols, copy resB on TMPA
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpA, AD_nrows, d_tmpA, HIP_R_64F));
        checkCudaErrors(hipMemcpyAsync(d_tmpA, d_resB, sizeof(double) * AD_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        // compute TMPB
        elem_mult_kernel<<<bSize, 32>>>(d_resXS, d_invS, d_tmpB, node.ncols);
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpB, AD_ncols, d_tmpB, HIP_R_64F));

        {    
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE TMPA = TMPA + A * TMPB
        {    
            alpha = 1.0;
            beta = 1.0;

            hipDeviceSynchronize();

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE DEL_Y = SOLVE(ADA, TMPA)
        {
            checkCudaErrors(hipsolverSpDcsrlsvchol(node.cusolverSpHandle,
                                            ADA_nrows, ADA_nnz, matDescrGen,
                                            d_ADAvals, d_ADAoffs, d_ADAinds,
                                            d_tmpA,
                                            node.env->MEHROTRA_CHOL_TOL, reorder,
                                            d_delY, &singularity));
        }

        ///////////////             COMPUTE DEL_S = - AT * DEL_Y - resC
        checkCudaErrors(hipMemcpyAsync(d_delS, d_resC, sizeof(double) * AT_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        ///////////////             COMPUTE DEL_X = -TMP_B - D * DEL_S
        checkCudaErrors(hipMemcpyAsync(d_delX, d_tmpB, sizeof(double) * D_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        // printf("%d) CORRECTION SYSTEM\n", iterations);
        // printf("delta X\n");
        // utils_printDvec(node.ncols, d_delX, true);
        // printf("delta Y\n");
        // utils_printDvec(node.nrows, d_delY, true);
        // printf("delta S\n");
        // utils_printDvec(node.ncols, d_delS, true);

        // finding alphaMaxPrim and alphaMaxDual: to improve
        find_alpha_max(&alphaMaxPrim, &alphaMaxDual,
                       d_x, d_delX, d_s, d_delS, node.ncols);

        alphaPrim = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxPrim);
        alphaDual = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxDual);

        // d_deltaX, d_deltaY, d_deltaS are pointees to d_sol
        // the solution of the previous system 

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaPrim, d_delX, 1, d_x, 1));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.nrows,
                                    &alphaDual, d_delY, 1, d_y, 1));
        
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaDual, d_delS, 1, d_s, 1));

        ///////////////             UPDATE

        alpha = -(alphaDual - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.ncols,
                                    &alpha, d_resC, 1));

        alpha = -(alphaPrim - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.nrows,
                                    &alpha, d_resB, 1));

        checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
        mu /= node.ncols;

        ++iterations;
    }

    node.iterations = iterations;
    
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols,
                               d_x, 1, node.d_ObjDns, 1, &node.objvalPrim));

    checkCudaErrors(hipblasDdot(node.cublasHandle, node.nrows,
                               d_y, 1, node.d_RhsDns, 1, &node.objvalDual));

    node.env->logger("Mehrotra procedure complete", "INFO", 10);
    node.timeSolverEnd = node.env->timer();

    ///////////////////             RELEASE MEMORY

    free(node.h_x);
    free(node.h_y);
    free(node.h_s);

    checkCudaErrors(hipFree(d_buffer1));
    checkCudaErrors(hipFree(d_buffer2));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_s));

    checkCudaErrors(hipFree(d_delX));
    checkCudaErrors(hipFree(d_delY));
    checkCudaErrors(hipFree(d_delS));
    
    checkCudaErrors(hipFree(d_resC));
    checkCudaErrors(hipFree(d_resB));
    checkCudaErrors(hipFree(d_resXS));
    
    checkCudaErrors(hipFree(d_invS));
    
    checkCudaErrors(hipsparseDestroyMatDescr(matDescrGen));

    hipsparseDestroySpMat(AT_descr);
    hipsparseDestroySpMat(D_descr);
    hipsparseDestroySpMat(AD_descr);
    hipsparseDestroySpMat(ADA_descr);

    checkCudaErrors(hipFree(d_AToffs));
    checkCudaErrors(hipFree(d_ATinds));
    checkCudaErrors(hipFree(d_ATvals));
    
    checkCudaErrors(hipFree(d_Dvals));
    checkCudaErrors(hipFree(d_Doffs));
    checkCudaErrors(hipFree(d_Dinds));

    checkCudaErrors(hipFree(d_ADvals));
    checkCudaErrors(hipFree(d_ADoffs));
    checkCudaErrors(hipFree(d_ADinds));

    checkCudaErrors(hipFree(d_ADAvals));
    checkCudaErrors(hipFree(d_ADAoffs));
    checkCudaErrors(hipFree(d_ADAinds));

    return CODE_SUCCESFULL;
}

SyphaStatus solver_sparse_mehrotra_3(SyphaNodeSparse &node)
{
    const int reorder = 0;
    int singularity = 0;

    int i = 0, j = 0, k = 0, iterations = 0;
    double alpha, beta, alphaPrim, alphaDual, sigma, mu, muAff;
    double alphaMaxPrim, alphaMaxDual;
    double *d_bufferX = NULL;
    double *d_bufferS = NULL;

    double *d_resC = NULL, *d_resB = NULL, *d_resXS = NULL;
    double *d_tmpA = NULL, *d_tmpB = NULL;
    double *d_x = NULL, *d_y = NULL, *d_s = NULL, *d_invS = NULL;
    double *d_delX = NULL, *d_delY = NULL, *d_delS = NULL;

    char message[1024];

    hipsparseDnVecDescr_t vecX, vecY, vecS, vecResC, vecResB;
    hipsparseDnVecDescr_t vecDelX, vecDelY, vecDelS, vecTmpA, vecTmpB;
    hipsparseSpGEMMDescr_t spgemmDescr;

    hipsparseMatDescr_t matDescrGen;
    checkCudaErrors(hipsparseCreateMatDescr(&matDescrGen));
    checkCudaErrors(hipsparseSetMatType(matDescrGen, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(matDescrGen, HIPSPARSE_INDEX_BASE_ZERO));

    // AT
    int64_t AT_nrows = node.ncols, AT_ncols = node.nrows, AT_nnz = node.nnz;
    int *d_AToffs = NULL, *d_ATinds = NULL;
    double *d_ATvals = NULL;
    hipsparseSpMatDescr_t AT_descr;

    checkCudaErrors(hipMalloc((void **)&d_AToffs, sizeof(int) * (AT_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&d_ATinds, sizeof(int) * AT_nnz));
    checkCudaErrors(hipMalloc((void **)&d_ATvals, sizeof(double) * AT_nnz));

    checkCudaErrors(hipsparseCreateCsr(&AT_descr, AT_nrows, AT_ncols, AT_nnz,
                                      d_AToffs, d_ATinds, d_ATvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // D
    int64_t D_nrows = node.ncols, D_ncols = node.ncols, D_nnz = node.ncols;
    int *d_Doffs = NULL, *d_Dinds = NULL;
    double *d_Dvals = NULL;
    hipsparseSpMatDescr_t D_descr;

    checkCudaErrors(hipMalloc((void **)&d_Doffs, sizeof(int) * (D_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&d_Dinds, sizeof(int) * D_nrows));
    checkCudaErrors(hipMalloc((void **)&d_Dvals, sizeof(double) * D_nrows));

    checkCudaErrors(hipsparseCreateCsr(&D_descr, D_nrows, D_ncols, D_nnz,
                                      d_Doffs, d_Dinds, d_Dvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // AD
    int64_t AD_nrows = node.nrows, AD_ncols = node.ncols, AD_nnz = 0, AD_currNnz = 0;
    int *d_ADoffs = NULL, *d_ADinds = NULL;
    double *d_ADvals = NULL;
    hipsparseSpMatDescr_t AD_descr;

    checkCudaErrors(hipMalloc((void **)&d_ADoffs, sizeof(int) * (AD_nrows + 1)));

    checkCudaErrors(hipsparseCreateCsr(&AD_descr, AD_nrows, AD_ncols, AD_nnz,
                                      d_ADoffs, d_ADinds, d_ADvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // ADA
    int64_t ADA_nrows = node.nrows, ADA_ncols = node.nrows, ADA_nnz = 0, ADA_currNnz = 0;
    int *d_ADAoffs = NULL, *d_ADAinds = NULL;
    double *d_ADAvals = NULL;
    hipsparseSpMatDescr_t ADA_descr;

    checkCudaErrors(hipMalloc((void **)&d_ADAoffs, sizeof(int) * (ADA_nrows + 1)));
    
    checkCudaErrors(hipsparseCreateCsr(&ADA_descr, ADA_nrows, ADA_ncols, ADA_nnz,
                                      d_ADAoffs, d_ADAinds, d_ADAvals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // BUFFER
    size_t bufferSize1 = 0;
    size_t currBufferSize1 = (size_t)(sizeof(double) * node.ncols * 2);
    double *d_buffer1 = NULL;

    size_t bufferSize2 = 0;
    size_t currBufferSize2 = 0;
    double *d_buffer2 = NULL;

    checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));

    ///////////////////             GET TRANSPOSED MATRIX

    checkCudaErrors(hipsparseCsr2cscEx2_bufferSize(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
                                                  node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
                                                  d_ATvals, d_AToffs, d_ATinds,
                                                  HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
                                                  HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
                                                  &bufferSize1));
    
    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseCsr2cscEx2(node.cusparseHandle, node.nrows, node.ncols, node.nnz,
                                       node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
                                       d_ATvals, d_AToffs, d_ATinds,
                                       HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
                                       HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG2,
                                       d_buffer1));

    ///////////////////             GET STARTING POINT
    // initialise x, y, s
    node.h_x = (double *)malloc(sizeof(double) * node.ncols);
    node.h_y = (double *)malloc(sizeof(double) * node.nrows);
    node.h_s = (double *)malloc(sizeof(double) * node.ncols);

    node.timeStartSolStart = node.env->timer();
    solver_sparse_mehrotra_init_gsl(node);
    node.timeStartSolEnd = node.env->timer();

    ///////////////////             INITIALISE RHS

    node.env->logger("Initialise right-hand-side", "INFO", 17);
    node.timePreSolStart = node.env->timer();

    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_y, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_s, sizeof(double) * node.ncols));

    checkCudaErrors(hipMalloc((void **)&d_delX, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_delY, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_delS, sizeof(double) * node.ncols));

    checkCudaErrors(hipMemcpyAsync(d_x, node.h_x, sizeof(double) * node.ncols, hipMemcpyHostToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_y, node.h_y, sizeof(double) * node.nrows, hipMemcpyHostToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_s, node.h_s, sizeof(double) * node.ncols, hipMemcpyHostToDevice, node.cudaStream));

    // put OBJ and S on device rhs
    checkCudaErrors(hipMalloc((void **)&d_resC, sizeof(double) * node.ncols));
    checkCudaErrors(hipMalloc((void **)&d_resB, sizeof(double) * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_resXS, sizeof(double) * node.ncols));

    checkCudaErrors(hipMemcpyAsync(d_resC, node.d_ObjDns, sizeof(double) * node.ncols, hipMemcpyDeviceToDevice, node.cudaStream));
    checkCudaErrors(hipMemcpyAsync(d_resB, node.d_RhsDns, sizeof(double) * node.nrows, hipMemcpyDeviceToDevice, node.cudaStream));

    // Residuals
    // resB, resC equation 14.7, page 395(414)Numerical Optimization
    // resC = AT * Y - (obj - s)
    // resB = A  * X - rhs

    checkCudaErrors(hipsparseCreateDnVec(&vecX, (int64_t)node.ncols, d_x, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecY, (int64_t)node.nrows, d_y, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecS, (int64_t)node.ncols, d_s, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecDelX, (int64_t)node.ncols, d_delX, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecDelY, (int64_t)node.nrows, d_delY, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecDelS, (int64_t)node.ncols, d_delS, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecResC, (int64_t)node.ncols, d_resC, HIP_R_64F));
    checkCudaErrors(hipsparseCreateDnVec(&vecResB, (int64_t)node.nrows, d_resB, HIP_R_64F));

    alpha = -1.0;
    checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                &alpha, d_s, 1, d_resC, 1));

    alpha = 1.0;
    beta = -1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, AT_descr, vecY,
                                            &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize1));

    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, AT_descr, vecY,
                                 &beta, vecResC, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 d_buffer1));

    alpha = 1.0;
    beta = -1.0;
    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            &alpha, node.matDescr, vecX,
                                            &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize1));

    if (bufferSize1 > currBufferSize1)
    {
        currBufferSize1 = bufferSize1;
        if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
        checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
    }

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, node.matDescr, vecX,
                                 &beta, vecResB, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 (size_t *)d_buffer1));

    ///////////////////             CALCULATE MU
    // duality measure, defined at page 395(414) Numerical Optimization
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
    mu /= node.ncols;

    node.timePreSolEnd = node.env->timer();

    ///////////////////             SET UP D AND INV(S)
    
    int bSize = (node.ncols >> 5) + 1;

    checkCudaErrors(hipMalloc((void **)&d_invS, sizeof(double) * node.ncols));
    
    hipDeviceSynchronize();
    range_kernel<<<bSize, 32>>>(d_Doffs, node.ncols + 1);
    range_kernel<<<bSize, 32>>>(d_Dinds, node.ncols);
    
    ///////////////////             MAIN LOOP
    
    node.env->logger("Starting Mehrotra proceduce", "INFO", 17);
    node.timeSolverStart = node.env->timer();
    
    iterations = 0;
    while ((iterations < node.env->MEHROTRA_MAX_ITER) && (mu > node.env->MEHROTRA_MU_TOL))
    {
        // x, s multiplication and res XS update: to improve
        hipDeviceSynchronize();
        elem_mult_kernel<<<bSize, 32>>>(d_x, d_s, d_resXS, node.ncols);

        elem_inv_kernel<<<bSize, 32>>>(d_s, d_invS, node.ncols);

        hipDeviceSynchronize();
        elem_mult_kernel<<<bSize, 32>>>(d_x, d_invS, d_Dvals, node.ncols);

        // hipDeviceSynchronize();
        // printf("%d) START\n", iterations);
        // printf("X\n");
        // utils_printDvec(node.ncols, d_x, true);
        // printf("INV(S)\n");
        // utils_printDvec(node.ncols, d_invS, true);
        // printf("D\n");
        // utils_printDvec(node.ncols, d_Dvals, true);
        // printf("RES B\n");
        // utils_printDvec(node.nrows, d_resB, true);
        // printf("RES C\n");
        // utils_printDvec(node.ncols, d_resC, true);

        ///////////////             COMPUTE AD
        {
            alpha = 1.0;
            beta = 0.0;

            // SpGEMM Computation
            checkCudaErrors(hipsparseSpGEMM_createDescr(&spgemmDescr));

            // ask bufferSize1 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, D_descr,
                                                  &beta, AD_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, NULL));

            if (bufferSize1 > currBufferSize1)
            {
                currBufferSize1 = bufferSize1;
                if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
                checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
            }

            // inspect the matrices D and AT to understand the memory requiremnent for
            // the next step
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, D_descr,
                                                  &beta, AD_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, d_buffer1));

            // ask bufferSize2 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, D_descr,
                                           &beta, AD_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, NULL));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            // compute the intermediate product of A * B
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, D_descr,
                                           &beta, AD_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, d_buffer2));

            // get matrix DA non-zero entries
            hipsparseSpMatGetSize(AD_descr, &AD_nrows, &AD_ncols, &AD_nnz);

            // allocate matrix DA
            if (AD_nnz > AD_currNnz)
            {
                AD_currNnz = AD_nnz;
                if (d_ADinds) checkCudaErrors(hipFree(d_ADinds));
                if (d_ADvals) checkCudaErrors(hipFree(d_ADvals));
                checkCudaErrors(hipMalloc((void **)&d_ADinds, sizeof(int) * AD_currNnz));
                checkCudaErrors(hipMalloc((void **)&d_ADvals, sizeof(double) * AD_currNnz));
            }

            // update DA with the new pointers
            checkCudaErrors(hipsparseCsrSetPointers(AD_descr, d_ADoffs, d_ADinds, d_ADvals));

            // copy the final products to the matrix AAT
            checkCudaErrors(hipsparseSpGEMM_copy(node.cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, node.matDescr, D_descr,
                                        &beta, AD_descr,
                                        HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDescr));

            checkCudaErrors(hipsparseSpGEMM_destroyDescr(spgemmDescr));
        }

        ///////////////             COMPUTE ADA
        {
            alpha = 1.0;
            beta = 0.0;

            // SpGEMM Computation
            checkCudaErrors(hipsparseSpGEMM_createDescr(&spgemmDescr));

            // ask bufferSize1 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, AD_descr, AT_descr,
                                                  &beta, ADA_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, NULL));

            if (bufferSize1 > currBufferSize1)
            {
                currBufferSize1 = bufferSize1;
                if (d_buffer1) checkCudaErrors(hipFree(d_buffer1));
                checkCudaErrors(hipMalloc((void **)&d_buffer1, currBufferSize1));
            }

            // inspect the matrices A and DA to understand the memory requiremnent for
            // the next step
            checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, AD_descr, AT_descr,
                                                  &beta, ADA_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, d_buffer1));

            // ask bufferSize2 bytes for external memory
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, AD_descr, AT_descr,
                                           &beta, ADA_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, NULL));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            // compute the intermediate product of A * DA
            checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, AD_descr, AT_descr,
                                           &beta, ADA_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, d_buffer2));

            // get matrix ADA non-zero entries
            hipsparseSpMatGetSize(ADA_descr, &ADA_nrows, &ADA_ncols, &ADA_nnz);

            // allocate matrix ADA
            if (ADA_nnz > ADA_currNnz)
            {
                ADA_currNnz = ADA_nnz;
                if (d_ADAinds) checkCudaErrors(hipFree(d_ADAinds));
                if (d_ADAvals) checkCudaErrors(hipFree(d_ADAvals));
                checkCudaErrors(hipMalloc((void **)&d_ADAinds, sizeof(int) * ADA_currNnz));
                checkCudaErrors(hipMalloc((void **)&d_ADAvals, sizeof(double) * ADA_currNnz));
            }

            // update ADA with the new pointers
            checkCudaErrors(hipsparseCsrSetPointers(ADA_descr, d_ADAoffs, d_ADAinds, d_ADAvals));

            // copy the final products to the matrix ADA
            checkCudaErrors(hipsparseSpGEMM_copy(node.cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, AD_descr, AT_descr,
                                        &beta, ADA_descr,
                                        HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDescr));

            checkCudaErrors(hipsparseSpGEMM_destroyDescr(spgemmDescr));
        }

        ///////////////////             TEST
        // double *d_ADn = NULL;
        // checkCudaErrors(hipMalloc((void **)&d_ADn, sizeof(double) * ADA_nrows * ADA_ncols));

        // hipsparseMatDescr_t matGenDescr;
        // checkCudaErrors(hipsparseCreateMatDescr(&matGenDescr));
        // checkCudaErrors(hipsparseSetMatType(matGenDescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
        // checkCudaErrors(hipsparseSetMatIndexBase(matGenDescr, HIPSPARSE_INDEX_BASE_ZERO));

        // checkCudaErrors(hipsparseDcsr2dense(node.cusparseHandle, ADA_nrows, ADA_ncols,
        //                                    matGenDescr, // HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO
        //                                    d_ADAvals, d_ADAoffs, d_ADAinds,
        //                                    d_ADn, ADA_nrows));

        // printf("%d) MAT\n", iterations);
        // utils_printDmat(ADA_nrows, ADA_ncols, ADA_nrows, d_ADn, true, false);
        // checkCudaErrors(hipFree(d_ADn));

        // printf("OFFS:\n");
        // utils_printIvec(A_nrows+1, d_csrAOffs, true);
        // printf("INDS:\n");
        // utils_printIvec(A_nnz, d_csrAInds, true);
        // printf("VALS:\n");
        // utils_printDvec(A_nnz, d_csrAVals, true);
        ///////////////////             END TEST

        ///////////////             COMPUTE TMPA = - AD * resC - resB

        // store TMPA vector on buffer 1, size of buffer 1 is guaranteed to 
        // be >= 2*ncols, copy resB on TMPA
        d_tmpA = d_buffer1;
        d_tmpB = &d_buffer1[node.ncols];
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpA, AD_nrows, d_tmpA, HIP_R_64F));
        checkCudaErrors(hipMemcpyAsync(d_tmpA, d_resB, sizeof(double) * AD_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        // compute TMPB
        elem_mult_kernel<<<bSize, 32>>>(d_resXS, d_invS, d_tmpB, node.ncols);
        checkCudaErrors(hipsparseCreateDnVec(&vecTmpB, AD_ncols, d_tmpB, HIP_R_64F));

        {    
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AD_descr, vecResC,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE TMPA = TMPA + A * TMPB
        {    
            alpha = 1.0;
            beta = 1.0;

            hipDeviceSynchronize();

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, node.matDescr, vecTmpB,
                        &beta, vecTmpA, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));
        }

        ///////////////             COMPUTE DEL_Y = SOLVE(ADA, TMPA)
        {
            checkCudaErrors(hipsolverSpDcsrlsvchol(node.cusolverSpHandle,
                                            ADA_nrows, ADA_nnz, matDescrGen,
                                            d_ADAvals, d_ADAoffs, d_ADAinds,
                                            d_tmpA,
                                            node.env->MEHROTRA_CHOL_TOL, reorder,
                                            d_delY, &singularity));
        }

        ///////////////             COMPUTE DEL_S = - AT * DEL_Y - resC
        checkCudaErrors(hipMemcpyAsync(d_delS, d_resC, sizeof(double) * AT_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, AT_descr, vecDelY,
                        &beta, vecDelS, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        ///////////////             COMPUTE DEL_X = -TMP_B - D * DEL_S
        checkCudaErrors(hipMemcpyAsync(d_delX, d_tmpB, sizeof(double) * D_nrows, hipMemcpyDeviceToDevice, node.cudaStream));

        {
            alpha = -1.0;
            beta = -1.0;

            // buffer 1 used for TMPA and TMPB
            checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        &bufferSize2));

            if (bufferSize2 > currBufferSize2)
            {
                currBufferSize2 = bufferSize2;
                if (d_buffer2) checkCudaErrors(hipFree(d_buffer2));
                checkCudaErrors(hipMalloc((void **)&d_buffer2, currBufferSize2));
            }

            checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, D_descr, vecDelS,
                        &beta, vecDelX, HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                        d_buffer2));   
        }

        // printf("%d) AFFINE SYSTEM\n", iterations);
        // printf("delta X\n");
        // utils_printDvec(node.ncols, d_delX, true);
        // printf("delta Y\n");
        // utils_printDvec(node.nrows, d_delY, true);
        // printf("delta S\n");
        // utils_printDvec(node.ncols, d_delS, true);

        // affine step length, definition 14.32 at page 408(427)
        // alpha_max_p = min([-xi / delta_xi for xi, delta_xi in zip(x, delta_x_aff) if delta_xi < 0.0])
        // alpha_max_d = min([-si / delta_si for si, delta_si in zip(s, delta_s_aff) if delta_si < 0.0])

        // finding alphaMaxPrim and alphaMaxDual: to improve
        find_alpha_max(&alphaMaxPrim, &alphaMaxDual,
                       d_x, d_delX, d_s, d_delS, node.ncols);

        alphaPrim = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxPrim);
        alphaDual = gsl_min(1.0, node.env->MEHROTRA_ETA * alphaMaxDual);

        // d_deltaX, d_deltaY, d_deltaS are pointees to d_sol
        // the solution of the previous system 

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaPrim, d_delX, 1, d_x, 1));

        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.nrows,
                                    &alphaDual, d_delY, 1, d_y, 1));
        
        checkCudaErrors(hipblasDaxpy(node.cublasHandle, node.ncols,
                                    &alphaDual, d_delS, 1, d_s, 1));

        ///////////////             UPDATE

        alpha = -(alphaDual - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.ncols,
                                    &alpha, d_resC, 1));

        alpha = -(alphaPrim - 1.0);
        checkCudaErrors(hipblasDscal(node.cublasHandle, node.nrows,
                                    &alpha, d_resB, 1));

        checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols, d_x, 1, d_s, 1, &mu));
        mu /= node.ncols;

        ++iterations;
    }

    node.iterations = iterations;
    
    checkCudaErrors(hipblasDdot(node.cublasHandle, node.ncols,
                               d_x, 1, node.d_ObjDns, 1, &node.objvalPrim));

    checkCudaErrors(hipblasDdot(node.cublasHandle, node.nrows,
                               d_y, 1, node.d_RhsDns, 1, &node.objvalDual));

    node.env->logger("Mehrotra procedure complete", "INFO", 10);
    node.timeSolverEnd = node.env->timer();

    ///////////////////             RELEASE MEMORY

    free(node.h_x);
    free(node.h_y);
    free(node.h_s);

    checkCudaErrors(hipFree(d_buffer1));
    checkCudaErrors(hipFree(d_buffer2));
    
    checkCudaErrors(hipFree(d_x));
    checkCudaErrors(hipFree(d_y));
    checkCudaErrors(hipFree(d_s));

    checkCudaErrors(hipFree(d_delX));
    checkCudaErrors(hipFree(d_delY));
    checkCudaErrors(hipFree(d_delS));
    
    checkCudaErrors(hipFree(d_resC));
    checkCudaErrors(hipFree(d_resB));
    checkCudaErrors(hipFree(d_resXS));
    
    checkCudaErrors(hipFree(d_invS));
    
    checkCudaErrors(hipsparseDestroyMatDescr(matDescrGen));

    hipsparseDestroySpMat(AT_descr);
    hipsparseDestroySpMat(D_descr);
    hipsparseDestroySpMat(AD_descr);
    hipsparseDestroySpMat(ADA_descr);

    checkCudaErrors(hipFree(d_AToffs));
    checkCudaErrors(hipFree(d_ATinds));
    checkCudaErrors(hipFree(d_ATvals));
    
    checkCudaErrors(hipFree(d_Dvals));
    checkCudaErrors(hipFree(d_Doffs));
    checkCudaErrors(hipFree(d_Dinds));

    checkCudaErrors(hipFree(d_ADvals));
    checkCudaErrors(hipFree(d_ADoffs));
    checkCudaErrors(hipFree(d_ADinds));

    checkCudaErrors(hipFree(d_ADAvals));
    checkCudaErrors(hipFree(d_ADAoffs));
    checkCudaErrors(hipFree(d_ADAinds));

    return CODE_SUCCESFULL;
}

SyphaStatus solver_sparse_mehrotra_init_1(SyphaNodeSparse &node)
{
    const int reorder = 0;
    int singularity = 0;

    int64_t AAT_nrows = node.nrows, AAT_ncols = node.nrows, AAT_nnz = 0;
    double alpha = 1.0;
    double beta = 0.0;

    int *AAT_inds = NULL, *AAT_offs = NULL;
    double *AAT_vals = NULL;

    void *d_buffer1 = NULL, *d_buffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    hipsparseSpMatDescr_t AAT_descr;
    hipsparseMatDescr_t AAT_descrGen, matTransDescrGen;
    hipsparseSpGEMMDescr_t spgemmDescr;

    checkCudaErrors(hipsparseCreateMatDescr(&AAT_descrGen));
    checkCudaErrors(hipsparseSetMatType(AAT_descrGen, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(AAT_descrGen, HIPSPARSE_INDEX_BASE_ZERO));

    checkCudaErrors(hipsparseCreateMatDescr(&matTransDescrGen));
    checkCudaErrors(hipsparseSetMatType(matTransDescrGen, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(matTransDescrGen, HIPSPARSE_INDEX_BASE_ZERO));

    ///////////////////             COMPUTE STARTING COORDINATES X AND S

    // AAT matrix for geMM
    checkCudaErrors(hipsparseCreateCsr(&AAT_descr, AAT_nrows, AAT_ncols, AAT_nnz,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

    // SpGEMM Computation
    checkCudaErrors(hipsparseSpGEMM_createDescr(&spgemmDescr));

    // ask bufferSize1 bytes for external memory
    checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, node.matTransDescr,
                                                  &beta, AAT_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, NULL));

    checkCudaErrors(hipMalloc((void **)&d_buffer1, bufferSize1));

    // inspect the matrices A and B to understand the memory requiremnent for
    // the next step
    checkCudaErrors(hipsparseSpGEMM_workEstimation(node.cusparseHandle,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                  &alpha, node.matDescr, node.matTransDescr,
                                                  &beta, AAT_descr,
                                                  HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                                  spgemmDescr, &bufferSize1, d_buffer1));

    // ask bufferSize2 bytes for external memory
    checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, node.matTransDescr,
                                           &beta, AAT_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, NULL));

    checkCudaErrors(hipMalloc((void **)&d_buffer2, bufferSize2));

    // compute the intermediate product of A * B
    checkCudaErrors(hipsparseSpGEMM_compute(node.cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha, node.matDescr, node.matTransDescr,
                                           &beta, AAT_descr,
                                           HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDescr, &bufferSize2, d_buffer2));

    // get matrix C non-zero entries C_num_nnz1
    //hipsparseSpMatGetSize(AAT_descr, &AAT_nrows, &AAT_ncols, &AAT_nnz);
    hipsparseSpMatGetSize(AAT_descr, &AAT_nrows, &AAT_ncols, &AAT_nnz);

    // allocate matrix AAT
    checkCudaErrors(hipMalloc((void **)&AAT_offs, sizeof(int) * (AAT_nrows + 1)));
    checkCudaErrors(hipMalloc((void **)&AAT_inds, sizeof(int) * AAT_nnz));
    checkCudaErrors(hipMalloc((void **)&AAT_vals, sizeof(double) * AAT_nnz));

    // update AAT with the new pointers
    checkCudaErrors(hipsparseCsrSetPointers(AAT_descr, AAT_offs, AAT_inds, AAT_vals));

    // copy the final products to the matrix AAT
    checkCudaErrors(hipsparseSpGEMM_copy(node.cusparseHandle,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, node.matDescr, node.matTransDescr,
                                        &beta, AAT_descr,
                                        HIP_R_64F, HIPSPARSE_SPGEMM_DEFAULT, spgemmDescr));

    checkCudaErrors(hipsparseSpGEMM_destroyDescr(spgemmDescr));

    int64_t r, c, n;
    std::cout << "buff 1: " << bufferSize1 << ", buff 2: " << bufferSize2 << std::endl;

    hipsparseSpMatGetSize(node.matDescr, &r, &c, &n);
    std::cout << "\nMat" << std::endl;
    std::cout << "rows: " << r << ", cols: " << c << ", num nz: " << n << std::endl;
    hipsparseSpMatGetSize(node.matTransDescr, &r, &c, &n);
    std::cout << "\nTrans" << std::endl;
    std::cout << "rows: " << r << ", cols: " << c << ", num nz: " << n << std::endl;
    hipsparseSpMatGetSize(AAT_descr, &r, &c, &n);
    std::cout << "\nAAT" << std::endl;
    std::cout << "rows: " << r << ", cols: " << c << ", num nz: " << n << std::endl;

    /*void *d_b, *d_x;
    checkCudaErrors(hipMalloc((void **)&d_b, AAT_nrows*sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_x, AAT_nrows*sizeof(double)));

    checkCudaErrors(hipsolverSpDcsrlsvchol(
        node.cusolverSpHandle, AAT_nrows, AAT_nnz,
        AAT_descrGen, AAT_vals, AAT_offs, AAT_inds,
        (double*)d_b, node.env->MEHROTRA_CHOL_TOL, reorder, (double*)d_x, &singularity));

    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_x));*/

    hipsparseSpMatGetSize(AAT_descr, &r, &c, &n);
    std::cout << "\nAAT" << std::endl;
    std::cout << "rows: " << r << ", cols: " << c << ", num nz: " << n << std::endl;

    ///////////////////             COMPUTE s = - mat' * y + obj
    alpha = -1.0;
    beta = 1.0;

    // copy obj on s
    checkCudaErrors(hipMemcpyAsync(node.d_s, node.d_ObjDns, sizeof(double) * node.ncols,
                                    hipMemcpyDeviceToDevice, node.cudaStream));

    checkCudaErrors(cusparseCsrmvEx_bufferSize(node.cusparseHandle, CUSPARSE_ALG_MERGE_PATH,
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               node.ncols, node.nrows, node.nnz,
                                               &alpha, HIP_R_64F,
                                               matTransDescrGen,
                                               node.d_csrMatTransVals, HIP_R_64F,
                                               node.d_csrMatTransOffs,
                                               node.d_csrMatTransInds,
                                               node.d_y, HIP_R_64F,
                                               &beta, HIP_R_64F,
                                               node.d_s, HIP_R_64F, HIP_R_64F,
                                               &bufferSize1));

    checkCudaErrors(hipMalloc((void **)&d_buffer1, bufferSize1));
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(cusparseCsrmvEx(node.cusparseHandle, CUSPARSE_ALG_MERGE_PATH,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    node.ncols, node.nrows, node.nnz,
                                    &alpha, HIP_R_64F,
                                    matTransDescrGen,
                                    node.d_csrMatTransVals, HIP_R_64F,
                                    node.d_csrMatTransOffs,
                                    node.d_csrMatTransInds,
                                    node.d_y, HIP_R_64F,
                                    &beta, HIP_R_64F,
                                    node.d_s, HIP_R_64F, HIP_R_64F,
                                    d_buffer1));

    checkCudaErrors(hipsparseDestroyMatDescr(AAT_descrGen));
    checkCudaErrors(hipsparseDestroyMatDescr(matTransDescrGen));
    checkCudaErrors(hipsparseDestroySpMat(AAT_descr));

    checkCudaErrors(hipFree(d_buffer1));
    checkCudaErrors(hipFree(d_buffer2));

    checkCudaErrors(hipFree(AAT_inds));
    checkCudaErrors(hipFree(AAT_offs));
    checkCudaErrors(hipFree(AAT_vals));

    return CODE_SUCCESFULL;
}

SyphaStatus solver_sparse_mehrotra_init_2(SyphaNodeSparse &node)
{
    const int reorder = 0;
    int singularity = 0;
    int info = 0;
    int i = 0;
    int I_matBytes = node.nrows * node.nrows * sizeof(double);

    double alpha = 1.0;
    double beta = 0.0;

    int *d_ipiv = NULL;
    double *d_AAT = NULL;
    double *d_matDn = NULL;
    double *h_I = NULL;

    void *d_buffer = NULL;
    size_t currBufferSize = 0;
    size_t bufferSize = 0;
    char message[1024];

    hipsolverDnParams_t cusolverDnParams;
    hipsparseDnVecDescr_t vecX, vecY, vecS;
    hipsparseDnMatDescr_t AAT_descr, matDnDescr;
    hipsparseMatDescr_t matDescrGen;

    node.env->logger("Mehrotra starting point computation", "INFO", 13);
    checkCudaErrors(hipsolverDnCreateParams(&cusolverDnParams));

    checkCudaErrors(hipsparseCreateMatDescr(&matDescrGen));
    checkCudaErrors(hipsparseSetMatType(matDescrGen, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(matDescrGen, HIPSPARSE_INDEX_BASE_ZERO));

    checkCudaErrors(hipsparseCreateDnVec(&vecX, (int64_t)node.ncols,
                                        node.d_x, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecY, (int64_t)node.nrows,
                                        node.d_y, HIP_R_64F));

    checkCudaErrors(hipsparseCreateDnVec(&vecS, (int64_t)node.ncols,
                                        node.d_s, HIP_R_64F));

    checkCudaErrors(hipMalloc((void **)&d_AAT, sizeof(double) * node.nrows * node.nrows));
    checkCudaErrors(hipMalloc((void **)&d_matDn, sizeof(double) * node.nrows * node.ncols));

    checkCudaErrors(hipsparseCreateDnMat(&AAT_descr, (int64_t)node.nrows, (int64_t)node.nrows,
                                        (int64_t)node.nrows, d_AAT, HIP_R_64F,
                                        HIPSPARSE_ORDER_COL));

    checkCudaErrors(hipsparseCreateDnMat(&matDnDescr, (int64_t)node.nrows, (int64_t)node.ncols,
                                        (int64_t)node.nrows, d_matDn, HIP_R_64F,
                                        HIPSPARSE_ORDER_COL));

    ///////////////////             STORE MATRIX IN DENSE FORMAT
    node.env->logger("solver_sparse_mehrotra_init - storing matrix in dense format", "INFO", 20);
    checkCudaErrors(hipsparseDcsr2dense(node.cusparseHandle, node.nrows, node.ncols,
                                       matDescrGen, // HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ZERO
                                       node.d_csrMatVals, node.d_csrMatOffs, node.d_csrMatInds,
                                       d_matDn, node.nrows));

    ///////////////////             COMPUTE AAT INVERSE MATRIX

    // GEMM Computation: MATRIX * MATRIX'
    node.env->logger("solver_sparse_mehrotra_init - computing mat * mat'", "INFO", 20);
    checkCudaErrors(hipsparseSpMM_bufferSize(node.cusparseHandle,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            HIPSPARSE_OPERATION_TRANSPOSE,
                                            &alpha, node.matDescr, matDnDescr,
                                            &beta, AAT_descr,
                                            HIP_R_64F,
                                            HIPSPARSE_CSRMM_ALG1,
                                            &bufferSize));

    // allocate memory for computation
    currBufferSize = bufferSize > I_matBytes ? bufferSize : I_matBytes;
    checkCudaErrors(hipMalloc((void **)&d_buffer, currBufferSize));

    checkCudaErrors(hipsparseSpMM(node.cusparseHandle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_TRANSPOSE,
                                 &alpha, node.matDescr, matDnDescr,
                                 &beta, AAT_descr,
                                 HIP_R_64F,
                                 HIPSPARSE_CSRMM_ALG1,
                                 d_buffer));

    ///////////////////             MATRIX INVERSION

    node.env->logger("solver_sparse_mehrotra_init - computing matrix inversion", "INFO", 20);
    // See https://stackoverflow.com/questions/50892906/what-is-the-most-efficient-way-to-compute-the-inverse-of-a-general-matrix-using
    checkCudaErrors(hipsolverDnDgetrf_bufferSize(node.cusolverDnHandle,
                                                node.nrows, node.nrows,
                                                d_AAT, node.nrows,
                                                (int *)&bufferSize));

    // allocate memory for computation
    if (bufferSize > currBufferSize)
    {
        currBufferSize = bufferSize;
        checkCudaErrors(hipMalloc((void **)&d_buffer, currBufferSize));
    }
    checkCudaErrors(hipMalloc((void **)&d_ipiv, sizeof(int) * node.nrows));

    /*checkCudaErrors(hipsolverDnDgetrf(node.cusolverDnHandle,
                                     node.nrows, node.nrows,
                                     d_AAT, node.nrows,
                                     (double *)d_buffer, d_ipiv,
                                     &info));*/

    printf("AAT after getrf\n");
    utils_printDmat(node.nrows, node.nrows, node.nrows, d_AAT, true, true);

    sprintf(message, "solver_sparse_mehrotra_init - cusolverDnGetrf returned %d", info);
    node.env->logger(message, "INFO", 20);

    // set I matrix
    h_I = (double *)calloc(node.nrows * node.nrows, sizeof(double));
    for (i = 0; i < node.nrows; ++i)
    {
        h_I[node.nrows * i + i] = 1.0;
    }
    //checkCudaErrors(hipMemcpyAsync(d_buffer, h_I, sizeof(double) * node.nrows * node.nrows, hipMemcpyHostToDevice, node.cudaStream));
    //checkCudaErrors(hipMemcpy(d_buffer, h_I, sizeof(double) * node.nrows * node.nrows, hipMemcpyHostToDevice));
    free(h_I);

    checkCudaErrors(hipsolverDnDgetrs(node.cusolverDnHandle, HIPBLAS_OP_N,
                                     node.nrows, node.nrows,
                                     d_AAT, node.nrows,
                                     d_ipiv,
                                     (double *)d_buffer, node.nrows,
                                     &info));

    printf("AAT after getrs\n");
    utils_printDmat(node.nrows, node.nrows, node.nrows, d_AAT, true, true);

    sprintf(message, "solver_sparse_mehrotra_init - cusolverDnGetrs returned %d", info);
    node.env->logger(message, "INFO", 20);

    /*void *d_b, *d_x;
    checkCudaErrors(hipMalloc((void **)&d_b, AAT_nrows*sizeof(double)));
    checkCudaErrors(hipMalloc((void **)&d_x, AAT_nrows*sizeof(double)));

    checkCudaErrors(hipsolverSpDcsrlsvchol(
        node.cusolverSpHandle, AAT_nrows, AAT_nnz,
        matDescrGen, AAT_vals, AAT_offs, AAT_inds,
        (double*)d_b, node.env->MEHROTRA_CHOL_TOL, reorder, (double*)d_x, &singularity));

    checkCudaErrors(hipFree(d_b));
    checkCudaErrors(hipFree(d_x));*/

    ///////////////////             COMPUTE s = - mat' * y + obj
    node.env->logger("solver_sparse_mehrotra_init - computing s = - mat' * y + obj", "INFO", 20);
    alpha = -1.0;
    beta = 1.0;

    // copy obj on s
    checkCudaErrors(hipMemcpyAsync(node.d_s, node.d_ObjDns, sizeof(double) * node.ncols,
                                    hipMemcpyDeviceToDevice, node.cudaStream));

    checkCudaErrors(hipsparseSpMV_bufferSize(node.cusparseHandle,
                                            HIPSPARSE_OPERATION_TRANSPOSE,
                                            &alpha, node.matDescr, vecY,
                                            &beta, vecS,
                                            HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                            &bufferSize));

    if (bufferSize > currBufferSize)
    {
        currBufferSize = bufferSize;
        checkCudaErrors(hipMalloc((void **)&d_buffer, currBufferSize));
    }
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipsparseSpMV(node.cusparseHandle,
                                 HIPSPARSE_OPERATION_TRANSPOSE,
                                 &alpha, node.matDescr, vecY,
                                 &beta, vecS,
                                 HIP_R_64F, HIPSPARSE_CSRMV_ALG2,
                                 d_buffer));

    ///////////////////             FREE RESOURCES
    checkCudaErrors(hipsolverDnDestroyParams(cusolverDnParams));

    checkCudaErrors(hipsparseDestroyMatDescr(matDescrGen));
    checkCudaErrors(hipsparseDestroyDnMat(AAT_descr));

    checkCudaErrors(hipsparseDestroyDnVec(vecX));
    checkCudaErrors(hipsparseDestroyDnVec(vecY));
    checkCudaErrors(hipsparseDestroyDnVec(vecS));

    checkCudaErrors(hipFree(d_ipiv));
    checkCudaErrors(hipFree(d_buffer));

    checkCudaErrors(hipFree(d_AAT));
    checkCudaErrors(hipFree(d_matDn));

    return CODE_SUCCESFULL;
}

SyphaStatus solver_sparse_mehrotra_init_gsl(SyphaNodeSparse &node)
{
    int i, j;
    int signum = 0;
    double deltaX, deltaS, prod, sumX, sumS;
    char message[1024];

    gsl_vector *x = NULL;
    gsl_vector *y = NULL;
    gsl_vector *s = NULL;
    gsl_matrix *inv = NULL;
    gsl_matrix *mat = NULL;
    gsl_matrix *tmp = NULL;
    gsl_permutation *perm = NULL;

    x = gsl_vector_alloc((size_t)node.ncols);
    y = gsl_vector_alloc((size_t)node.nrows);
    s = gsl_vector_alloc((size_t)node.ncols);
    inv = gsl_matrix_calloc((size_t)node.nrows, (size_t)node.nrows);
    mat = gsl_matrix_calloc((size_t)node.nrows, (size_t)node.ncols);
    tmp = gsl_matrix_calloc((size_t)node.nrows, (size_t)node.ncols);
    perm = gsl_permutation_alloc((size_t)node.nrows);

    // csr to dense
    for (i = 0; i < node.nrows; ++i)
    {
        for (j = node.h_csrMatOffs->data()[i]; j < node.h_csrMatOffs->data()[i + 1]; ++j)
        {
            mat->data[node.ncols * i + node.h_csrMatInds->data()[j]] = node.h_csrMatVals->data()[j];
        }
    }
    //printf("MAT:\n");
    //utils_printDmat(node.nrows, node.ncols, node.ncols, mat->data, false);

    ///////////////////             MATRIX MULT
    node.env->logger("solver_sparse_mehrotra_init - computing A * A'", "INFO", 20);
    mat->size1 = node.nrows;
    mat->size2 = node.ncols;
    mat->tda = node.ncols;
    tmp->size1 = node.nrows;
    tmp->size2 = node.nrows;
    tmp->tda = node.ncols;
    gsl_blas_dgemm(CblasNoTrans, CblasTrans, 1.0, mat, mat, 0.0, tmp);

    //printf("AAT:\n");
    //utils_printDmat(node.nrows, node.nrows, node.ncols, tmp->data, false);

    ///////////////////             MATRIX INVERSION
    node.env->logger("solver_sparse_mehrotra_init - computing inv(AAT)", "INFO", 20);
    
    inv->size1 = node.nrows;
    inv->size2 = node.nrows;
    inv->tda = node.nrows;
    gsl_linalg_LU_decomp(tmp, perm, &signum);
    gsl_linalg_LU_invert(tmp, perm, inv);

    //printf("INV:\n");
    //utils_printDmat(node.nrows, node.nrows, node.nrows, inv->data, false);

    ///////////////////             COMPUTE x = mat' * AAT_inv * rhs
    node.env->logger("solver_sparse_mehrotra_init - computing x <-- A' * inv(AAT) * rhs", "INFO", 20);

    tmp->size1 = node.ncols;
    tmp->size2 = node.nrows;
    tmp->tda = node.nrows;
    gsl_blas_dgemm(CblasTrans, CblasNoTrans, 1.0, mat, inv, 0.0, tmp);

    // put RHS in Y
    memcpy(y->data, node.h_RhsDns, sizeof(double) * node.nrows);
    gsl_blas_dgemv(CblasNoTrans, 1.0, tmp, y, 0.0, x);

    //printf("TMP:\n");
    //utils_printDmat(node.ncols, node.nrows, node.nrows, tmp->data, false);

    ///////////////////             COMPUTE y = AAT_inv * mat * obj
    node.env->logger("solver_sparse_mehrotra_init - computing y <-- inv(AAT) * A * obj", "INFO", 20);

    tmp->size1 = node.nrows;
    tmp->size2 = node.ncols;
    tmp->tda = node.ncols;
    
    // put OBJ in S
    memcpy(s->data, node.h_ObjDns, sizeof(double) * node.ncols);
    
    gsl_blas_dgemm(CblasNoTrans, CblasNoTrans, 1.0, inv, mat, 0.0, tmp);
    gsl_blas_dgemv(CblasNoTrans, 1.0, tmp, s, 0.0, y);

    //printf("TMP:\n");
    //utils_printDmat(node.ncols, node.nrows, node.nrows, tmp->data, false);

    ///////////////////             COMPUTE s = - mat' * y + obj
    node.env->logger("solver_sparse_mehrotra_init - computing s <-- obj - A' * y", "INFO", 20);
    gsl_blas_dgemv(CblasTrans, -1.0, mat, y, 1.0, s);

    deltaX = gsl_max(-1.5 * gsl_vector_min(x), 0.0);
    deltaS = gsl_max(-1.5 * gsl_vector_min(s), 0.0);

    gsl_vector_add_constant(x, deltaX);
    gsl_vector_add_constant(s, deltaS);

    gsl_blas_ddot(x, s, &prod);
    prod *= 0.5;

    sumX = 0.0;
    sumS = 0.0;
    for (j = 0; j < node.ncols; ++j)
    {
        sumX += x->data[j];
        sumS += s->data[j];
    }
    deltaX = prod / sumS;
    deltaS = prod / sumX;

    gsl_vector_add_constant(x, deltaX);
    gsl_vector_add_constant(s, deltaS);

    //printf("X:\n");
    //utils_printDvec(node.ncols, x->data, false);
    //printf("Y:\n");
    //utils_printDvec(node.nrows, y->data, false);
    //printf("S:\n");
    //utils_printDvec(node.ncols, s->data, false);

    memcpy(node.h_x, x->data, sizeof(double) * node.ncols);
    memcpy(node.h_y, y->data, sizeof(double) * node.nrows);
    memcpy(node.h_s, s->data, sizeof(double) * node.ncols);

    gsl_vector_free(x);
    gsl_vector_free(y);
    gsl_vector_free(s);
    gsl_matrix_free(inv);
    gsl_matrix_free(mat);
    gsl_matrix_free(tmp);
    gsl_permutation_free(perm);

    return CODE_SUCCESFULL;
}
